#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include <chrono>
#include <iostream>

#include "../HandlingErrors/Assert.h"
#include "../VisualStudio/ScopedInstrument.h"

void DotArrays_ref(int count, float* r, const float* a, const float* b) {
    for (int i = 0; i < count; ++i) {
        const int j = i * 4;
        // treat each block of four floats as a 
        // single four-element vector
        r[i] = a[j + 0] * b[j + 0]
             + a[j + 1] * b[j + 1]
             + a[j + 2] * b[j + 2]
             + a[j + 3] * b[j + 3];
    }
}

void CmpArrays_ref(int count, const float* a, const float* b) {
    for (int i = 0; i < count; ++i) {
        if (a[i] != b[i]) {
            DEBUG_BREAK();
        }
    }
}

__global__ void CmpKernel_CUDA(int count, const float* a, const float* b) {

    int i = blockDim.x*blockIdx.x + threadIdx.x;
    
    if (i < count) {
        if (a[i] != b[i]) {
            DEBUG_BREAK();
        }
    }
}

__global__ void DotKernel_CUDA(float* r, const float* a, const float* b, unsigned int count) {

    // CUDA provides a thread index to each invocation
    // of the kernel, this is our loop i
    int i = blockDim.x*blockIdx.x + threadIdx.x;
    
    if (i < count) {
        // treat each block of four floats as a 
        // single four-element vector
        const unsigned j = i * 4;
        r[i] = a[j + 0] * b[j + 0]
             + a[j + 1] * b[j + 1]
             + a[j + 2] * b[j + 2]
             + a[j + 3] * b[j + 3];
    }
}

int main() {

    unsigned int SIZE = 16000000;
    unsigned int SIZE_R = SIZE/4;
    float* arrayA = new float[SIZE];
    float* arrayB = new float[SIZE];
    float* arrayC = new float[SIZE_R];
    float* arrayC_CUDA = new float[SIZE_R];

    hipError_t cudaStatus;

    for (int i = 0; i < SIZE; i++) {
        arrayA[i] = 10 * rand() / RAND_MAX;
        arrayB[i] = 10 * rand() / RAND_MAX;
    }

    // allocate managed buffers that are visible to both GPU and CPU
    float* cr, *cr_ref, * ca, * cb;
    {
        ScopedInstrument instr("\nTime spent to allocate CUDA managed resources and copy input buffers: ");

        hipSetDevice(0);

        hipMalloc((void**)&ca, SIZE * sizeof(float));
        hipMalloc((void**)&cb, SIZE * sizeof(float));
        hipMalloc((void**)&cr, SIZE_R * sizeof(float));
        hipMalloc((void**)&cr_ref, SIZE_R * sizeof(float));

        // transfer the data into GPU-visible memory
        hipMemcpy(ca, arrayA, SIZE * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(cb, arrayB, SIZE * sizeof(float), hipMemcpyHostToDevice);

    }
    std::cout << "\n=========================================================================================================\n";

    std::cout << "\n=========================================================================================================\n";
    std::cout << "\nTime spent to make dot product of 2 arrays of size: " << SIZE << "\n\n";

    // ref
    {
        ScopedInstrument instr("Serial execution: ");
        DotArrays_ref(SIZE_R, arrayC, arrayA, arrayB);
    }

    int threadsPerBlock = 128;
    int blocksPerGrid = (SIZE_R + threadsPerBlock - 1) / threadsPerBlock;

    // cuda
    {
        ScopedInstrument instr("CUDA execution: ");

        // run the kernel on the GP
        DotKernel_CUDA<<<blocksPerGrid, threadsPerBlock>>>(cr, ca, cb, SIZE_R);

        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            std::cout << hipGetErrorString(cudaStatus) << "\n";
            DEBUG_BREAK();
        }

        // wait for the GPU to finish
        cudaStatus = hipDeviceSynchronize();
        
        if (cudaStatus != hipSuccess) {
            std::cout << hipGetErrorString(cudaStatus) << "\n";
            DEBUG_BREAK();
        }
    }
        
    hipMemcpy(arrayC_CUDA, cr, SIZE_R * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(cr_ref, arrayC, SIZE_R * sizeof(float), hipMemcpyHostToDevice);

    // compare the two result buffers serially
    {
        ScopedInstrument instr("Serial comparison: ");
        CmpArrays_ref(SIZE_R, arrayC, arrayC_CUDA);
    }
    std::cout << "\n=========================================================================================================\n";

    // compare the two result buffers with cuda
    {
        ScopedInstrument instr("CUDA comparison: ");
        // run the kernel on the GP
        CmpKernel_CUDA<<<blocksPerGrid, threadsPerBlock>>>(SIZE_R, cr, cr_ref);

        // wait for the GPU to finish
        hipDeviceSynchronize();

    }
    std::cout << "\n=========================================================================================================\n";

    {
        std::cout << "\n=========================================================================================================\n";
        ScopedInstrument instr("\nTime spent to deallocate CUDA managed resources: ");

        hipFree(cr_ref);
        hipFree(cr);
        hipFree(ca);
        hipFree(cb);

    }
    std::cout << "\n=========================================================================================================\n";

    delete[] arrayA;
    delete[] arrayB;
    delete[] arrayC;
    delete[] arrayC_CUDA;

    return 0;
}
